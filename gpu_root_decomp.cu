#include "hip/hip_runtime.h"
////////////////////////////////////////////
// Decompress ROOT compressed files.
////////////////////////////////////////////

#include <random>
#include <assert.h>
#include <iostream>
#include <vector>
#include <unistd.h>
#include <string>
#include <chrono>

#include "utils.h"
#include "pack.cuh"
#include "RZip.h"
#include "TError.h"
#include "nvcomp/zstd.h"
#include "nvcomp/lz4.h"
#include "nvcomp/deflate.h"

using Clock = std::chrono::high_resolution_clock;

bool verbose = false;

// The size of the ROOT block framing headers for compression:
// - 3 bytes to identify the compression algorithm and version.
// - 3 bytes to identify the deflated buffer size.
// - 3 bytes to identify the inflated buffer size.
#define HDRSIZE 9

__global__ void PrintBatch(void **chunk_pointers, size_t *chunk_sizes, char *data, size_t nChunks)
{
   printf("------------------------- BATCH ----------------------------\n");
   printf("nChunks: %li\n", nChunks);

   printf("chunk_sizes:\n");
   size_t total_size = 0;
   for (int i = 0; i < nChunks; i++) {
      printf("%li ", chunk_sizes[i]);
      total_size += chunk_sizes[i];
   }
   printf("\n");

   printf("total_size: %li\n", total_size);

   printf("data:\n");
   for (int k = 0; k < 100; k++) {
      printf("%c ", data[k]);
   }
   printf("\n");

   printf("chunk_pointers:\n");
   for (int c = 0; c < nChunks; c++) {
      printf("\tchunk %d:\n\t\t", c);
      for (int j = 0; j < min(100, (int)chunk_sizes[c]); j++) {
         printf("%c ", ((char *)chunk_pointers[c])[j]);
      }
      printf("\n");
   }
   printf("\n");
}

__global__ void CheckStatuses(nvcompStatus_t *statusPtrs, size_t nChunks)
{
   for (int i = 0; i < nChunks; i++) {
      if (statusPtrs[i] != nvcompSuccess) {
         printf("Decompression of chunk %d has FAILED with status: %d\n", i, statusPtrs[i]);
      }
   }
}

struct Result {
   float setupTime, decompTime, unpackTime;
   std::vector<char> decompressed;
};

class GPUDecompressor {
private:
   hipStream_t stream;
   size_t nChunks;
   size_t compTotalSize, decompTotalSize;
   bool packed;

   // Host buffers
   std::vector<std::vector<char>> hCompressed;
   std::vector<char> hDecompressed;
   std::vector<size_t> hCompSizes, hDecompSizes;

   // Device buffers;
   char *dCompressed, *dDecompressed, *dUnpackOut;
   void **dCompressedChunkPointers, **dDecompressedChunkPointers;
   size_t *dCompSizes, *dDecompSizes;
   size_t tempBufSize;
   void *dTempBuf;
   nvcompStatus_t *dStatusPtrs;

   float setupTime, decompTime, unpackTime;

   inline std::vector<void *> GetCompressedChunkPtrs()
   {
      std::vector<void *> ptrs(nChunks);
      size_t offset = HDRSIZE;

      for (size_t i = 0; i < nChunks; ++i) {
         ptrs[i] = static_cast<void *>(dCompressed + offset);
         offset += hCompSizes[i];
      }
      return ptrs;
   }

   inline std::vector<void *> GetDecompressedChunkPtrs()
   {
      std::vector<void *> ptrs(nChunks);

      size_t offset = 0;
      for (size_t i = 0; i < nChunks; ++i) {
         ptrs[i] = static_cast<void *>(dDecompressed + offset);
         offset += hDecompSizes[i];
      }
      return ptrs;
   }

   // Allocate and setup various host/device buffers for decompressing the input data.
   template <typename GetDecompressSizeFunc, typename GetTempSizeFunc>
   void Configure(GetDecompressSizeFunc nvcompGetDecompressSize, GetTempSizeFunc nvcompGetDecompressTempSize)
   {
      // For measuring setup time
      auto configureStart = Clock::now();

      decompTotalSize = 0;
      int maxUncompressedChunkSize = 0;
      for (int i = 0; i < hCompressed.size(); i++) {
         size_t remainder = hCompressed[i].size();
         auto source = const_cast<unsigned char *>(reinterpret_cast<const unsigned char *>(hCompressed[i].data()));

         // Loop over the chunks to determine their sizes from the header
         do {
            int szSource;
            int szTarget;
            int retval = R__unzip_header(&szSource, source, &szTarget);
            R__ASSERT(retval == 0);
            R__ASSERT(szSource > 0);
            R__ASSERT(szTarget > szSource);
            R__ASSERT(static_cast<unsigned char>(szSource) <= hCompressed[i].size());

            nChunks++;
            hCompSizes.push_back(szSource);
            hDecompSizes.push_back(szTarget);

            decompTotalSize += szTarget;
            if (szTarget > maxUncompressedChunkSize)
               maxUncompressedChunkSize = szTarget;

            // Move to next chunk
            source += szSource;
            remainder -= szSource;
         } while (remainder > 0);
         R__ASSERT(remainder == 0);
      }

      hDecompressed.resize(decompTotalSize);

      // Set up buffers for the compressed and decompressed data on the device.
      ERRCHECK(hipMallocAsync(&dCompressed, compTotalSize * sizeof(char), stream));
      int offset = 0;
      for (int i = 0; i < hCompressed.size(); i++) {
         ERRCHECK(hipMemcpyAsync(&dCompressed[offset], hCompressed[i].data(), hCompressed[i].size() * sizeof(char),
                                  hipMemcpyHostToDevice, stream));
         offset += hCompressed[i].size();
      }
      ERRCHECK(hipMallocAsync(&dDecompressed, decompTotalSize * sizeof(char), stream));

      // Wait for the buffers to be allocated to create chunk pointers.
      ERRCHECK(hipStreamSynchronize(stream));

      // Set up pointers to each chunk in the device buffer for the compressed data.
      auto cPtrs = GetCompressedChunkPtrs();
      ERRCHECK(hipMallocAsync(&dCompressedChunkPointers, nChunks * sizeof(void *), stream));
      ERRCHECK(hipMemcpyAsync(dCompressedChunkPointers, cPtrs.data(), nChunks * sizeof(void *), hipMemcpyHostToDevice,
                               stream));

      // Set up pointers to each chunk in the device buffer for the decompressed data.
      auto dcPtrs = GetDecompressedChunkPtrs();
      ERRCHECK(hipMallocAsync(&dDecompressedChunkPointers, nChunks * sizeof(void *), stream));
      ERRCHECK(hipMemcpyAsync(dDecompressedChunkPointers, dcPtrs.data(), nChunks * sizeof(void *),
                               hipMemcpyHostToDevice, stream));

      // Copy compressed and decompressed sizes of each chunk
      std::transform(hCompSizes.begin(), hCompSizes.end(), hCompSizes.begin(), [&](auto x) { return x - HDRSIZE; });
      ERRCHECK(hipMallocAsync(&dCompSizes, hCompSizes.size() * sizeof(size_t), stream));
      ERRCHECK(hipMemcpyAsync(dCompSizes, hCompSizes.data(), hCompSizes.size() * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
      ERRCHECK(hipMallocAsync(&dDecompSizes, hDecompSizes.size() * sizeof(size_t), stream));
      ERRCHECK(hipMemcpyAsync(dDecompSizes, hDecompSizes.data(), hDecompSizes.size() * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));

      // Allocate temp space
      nvcompStatus_t status =
         nvcompGetDecompressTempSize(nChunks, maxUncompressedChunkSize, &tempBufSize, decompTotalSize);
      if (status != nvcompSuccess) {
         throw std::runtime_error("nvcompBatched*DecompressGetTempSize() failed.");
      }
      ERRCHECK(hipMallocAsync(&dTempBuf, tempBufSize, stream));

      // Status pointers
      ERRCHECK(hipMallocAsync(&dStatusPtrs, nChunks * sizeof(nvcompStatus_t), stream));

      if (packed) {
         ERRCHECK(hipMallocAsync(&dUnpackOut, decompTotalSize * sizeof(size_t), stream));
      }

      ERRCHECK(hipDeviceSynchronize());
      setupTime = std::chrono::duration_cast<std::chrono::nanoseconds>(Clock::now() - configureStart).count() / 1e6;

      if (verbose) {
         std::cout << "chunks        : " << nChunks << std::endl;
         PrintBatch<<<1, 1, 0, stream>>>(dCompressedChunkPointers, dCompSizes, dCompressed, nChunks);
         ERRCHECK(hipPeekAtLastError());
         PrintBatch<<<1, 1, 0, stream>>>(dDecompressedChunkPointers, dDecompSizes, dDecompressed, nChunks);
         ERRCHECK(hipPeekAtLastError());
      }
   }

   template <typename GetDecompressSizeFunc, typename GetTempSizeFunc, typename DecompressFunc>
   void DecompressInternal(GetDecompressSizeFunc nvcompGetDecompressSize, GetTempSizeFunc nvcompGetDecompressTempSize,
                           DecompressFunc nvcompDecompress)
   {
      Configure(nvcompGetDecompressSize, nvcompGetDecompressTempSize);

      // CUDA events to measure time
      hipEvent_t decompStart, decompEnd, unpackStart, unpackEnd;

      // For measuring decompression runtime
      ERRCHECK(hipEventCreate(&decompStart));
      ERRCHECK(hipEventCreate(&decompEnd));

      // Run decompression
      ERRCHECK(hipEventRecord(decompStart, stream));
      nvcompStatus_t status =
         nvcompDecompress(dCompressedChunkPointers, dCompSizes, dDecompSizes, dDecompSizes, nChunks, dTempBuf,
                          tempBufSize, dDecompressedChunkPointers, dStatusPtrs, stream);
      if (status != nvcompSuccess) {
         throw std::runtime_error("ERROR: nvcompBatched*DecompressAsync() not successful");
      }
      ERRCHECK(hipEventRecord(decompEnd, stream));
      ERRCHECK(hipEventSynchronize(decompEnd));
      ERRCHECK(hipEventElapsedTime(&decompTime, decompStart, decompEnd));
      ERRCHECK(hipEventDestroy(decompStart));
      ERRCHECK(hipEventDestroy(decompEnd));

      // Unpack data if necessary
      if (packed) {
         ERRCHECK(hipEventCreate(&unpackStart));
         ERRCHECK(hipEventCreate(&unpackEnd));
         ERRCHECK(hipEventRecord(unpackStart, stream));

         Unpack1<float, float><<<ceil(decompTotalSize / 256.), 256, 0, stream>>>(
            dUnpackOut, dDecompressed, dDecompSizes, nChunks, decompTotalSize);
         ERRCHECK(hipPeekAtLastError());
         dDecompressed = dUnpackOut;

         ERRCHECK(hipEventRecord(unpackEnd, stream));
         ERRCHECK(hipEventSynchronize(unpackEnd));
         ERRCHECK(hipEventElapsedTime(&unpackTime, unpackStart, unpackEnd));
         ERRCHECK(hipEventDestroy(unpackStart));
         ERRCHECK(hipEventDestroy(unpackEnd));
      }

      if (verbose) {
         CheckStatuses<<<1, 1, 0, stream>>>(dStatusPtrs, nChunks);
         ERRCHECK(hipPeekAtLastError());
         PrintBatch<<<1, 1, 0, stream>>>(dDecompressedChunkPointers, dDecompSizes, dDecompressed, nChunks);
         ERRCHECK(hipPeekAtLastError());
      }
   }

public:
   GPUDecompressor(const std::vector<std::vector<char>> &data, const size_t totalSize, bool _packed)
      : hCompressed(data), packed(_packed)
   {
      nChunks = 0;
      compTotalSize = totalSize;
      setupTime = 0;
      decompTime = 0;
      ERRCHECK(hipStreamCreate(&stream));
   }

   ~GPUDecompressor()
   {
      ERRCHECK(hipFree(dDecompressed));
      ERRCHECK(hipFree(dCompressed));
      ERRCHECK(hipFree(dCompressedChunkPointers));
      ERRCHECK(hipFree(dDecompressedChunkPointers));
      ERRCHECK(hipFree(dCompSizes));
      ERRCHECK(hipFree(dDecompSizes));
      ERRCHECK(hipFree(dTempBuf));
      ERRCHECK(hipFree(dStatusPtrs));
      ERRCHECK(hipStreamDestroy(stream));
   }

   bool Decompress(std::string type)
   {
      if (type == "zstd") {
         DecompressInternal(nvcompBatchedZstdGetDecompressSizeAsync, nvcompBatchedZstdDecompressGetTempSizeEx,
                            nvcompBatchedZstdDecompressAsync);
      } else if (type == "lz4") {
         DecompressInternal(nvcompBatchedLZ4GetDecompressSizeAsync, nvcompBatchedLZ4DecompressGetTempSizeEx,
                            nvcompBatchedLZ4DecompressAsync);
      } else if (type == "zlib") {
         DecompressInternal(nvcompBatchedDeflateGetDecompressSizeAsync, nvcompBatchedDeflateDecompressGetTempSizeEx,
                            nvcompBatchedDeflateDecompressAsync);
      } else {
         fprintf(stderr, "Unknown decompression type\n");
         return false;
      }

      return true;
   }

   Result GetResult()
   {
      Result result;

      // Retrieve resuts
      ERRCHECK(hipMemcpyAsync(hDecompressed.data(), dDecompressed, hDecompressed.size() * sizeof(char),
                               hipMemcpyDeviceToHost, stream));
      ERRCHECK(hipStreamSynchronize(stream));

      result.decompTime = decompTime;
      result.setupTime = setupTime;
      result.unpackTime = unpackTime;
      result.decompressed = hDecompressed;

      return result;
   }
};

/**
 * Main
 */

int main(int argc, char *argv[])
{
   std::string fileName, type, outputFile;
   int repetitions = 1;
   int multiFileSize = 1;
   int warmUp = 10;
   bool packed = false;

   int c;
   while ((c = getopt(argc, argv, "f:t:o:vn:m:w:p")) != -1) {
      switch (c) {
      case 'f': fileName = optarg; break;
      case 't': type = optarg; break;
      case 'o': outputFile = optarg; break;
      case 'v': verbose = true; break;
      case 'n': repetitions = atoi(optarg); break;
      case 'm': multiFileSize = atoi(optarg); break;
      case 'w': warmUp = atoi(optarg); break;
      case 'p': packed = true; break;
      default: std::cout << "Ignoring unknown parse returns: " << char(c) << std::endl; ;
      }
   }

   if (fileName.empty() || type.empty()) {
      std::cerr << "Must specify a file (-f) and decompression type (-t)" << std::endl;
      return 1;
   }

   auto files = GenerateMultiFile(fileName, multiFileSize);
   size_t totalSize = 0;
   for (int i = 0; i < files.size(); i++) {
      totalSize += files[i].size();
   }

   std::cout << "--------------------- INPUT INFORMATION ---------------------" << std::endl;
   std::cout << "file name      : " << fileName.c_str() << std::endl;
   std::cout << "type           : " << type.c_str() << std::endl;
   std::cout << "compressed (B) : " << totalSize << std::endl;
   std::cout << "repetitions    : " << repetitions << std::endl;
   std::cout << "warmup         : " << warmUp << std::endl;
   std::cout << "packed         : " << (packed ? "yes" : "no") << std::endl;

   std::vector<float> setupTimes, decompTimes, unpackTimes;
   Result result;
   for (int i = 0; i < repetitions + warmUp; i++) {
      GPUDecompressor decompressor(files, totalSize, packed);
      decompressor.Decompress(type);
      result = decompressor.GetResult();

      if (i >= warmUp) {
         setupTimes.push_back(result.setupTime);
         decompTimes.push_back(result.decompTime);
         unpackTimes.push_back(result.unpackTime);
      }
   }

   std::cout << "--------------------- OUTPUT INFORMATION ---------------------" << std::endl;
   std::cout << "decompressed (B): " << result.decompressed.size() << std::endl;
   std::cout << "Ratio\t\tAvg setup (ms)\tStdDev\t\tAvg decomp (ms)\t\tStdDev\t\tAvg unpack (ms)\t\tStdDev" << std::endl;
   std::cout << result.decompressed.size() / (double)totalSize << "\t\t" << GetMean(setupTimes) << "\t"
             << GetStdDev(setupTimes) << "\t\t" << GetMean(decompTimes) << "\t\t" << GetStdDev(decompTimes) << "\t\t"
             << GetMean(unpackTimes) << "\t" << GetStdDev(unpackTimes) << std::endl;

   if (!outputFile.empty()) {
      std::cout << "output file: " << outputFile.c_str() << std::endl;
      auto fp = fopen(outputFile.c_str(), "w");
      for (auto i = 0; i < result.decompressed.size(); i++) {
         fprintf(fp, "%c", result.decompressed[i]);
      }
   }

   return 0;
}
