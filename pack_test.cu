#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cassert>

#include "utils.h"
#include "pack.cuh"


int main(int argc, char const *argv[])
{
   std::vector<float> in(64000*10, 123456.);
   std::vector<float> pack(in.size());
   std::vector<float> out(in.size());
   std::vector<float> unpack(in.size());
   std::vector<size_t> sizes(10, 64000);
   // std::vector<size_t> sizes{256, 3, 1};
   // std::vector<size_t> sizes{1, 1, 1, 1};
   std::vector<size_t> chunkOffsets(sizes.size());

   // pack input buffer and unpack to test correctness
   size_t offset = 0;
   for (auto c = 0; c < sizes.size(); c++) {
      CastSplitPack<float, float>(&pack.data()[offset], &in.data()[offset], sizes[c]);
      offset += sizes[c];
   }

   offset = 0;
   for (auto c = 0; c < sizes.size(); c++) {
      CastSplitUnpack<float, float>(&unpack.data()[offset], &pack.data()[offset], sizes[c]);
      offset += sizes[c];
   }
   assert(in == unpack);

   // allocate device buffers
   float *dOut = NULL;
   ERRCHECK(hipMalloc(&dOut, out.size() * sizeof(float)));

   float *dPack = NULL;
   ERRCHECK(hipMalloc(&dPack, pack.size() * sizeof(float)));
   ERRCHECK(hipMemcpy(dPack, pack.data(), pack.size() * sizeof(float), hipMemcpyHostToDevice));

   size_t *dSizes = NULL;
   std::transform(sizes.begin(), sizes.end(), sizes.begin(), [&](auto &x) { return x * sizeof(float); });
   ERRCHECK(hipMalloc(&dSizes, sizes.size() * sizeof(size_t)));
   ERRCHECK(hipMemcpy(dSizes, sizes.data(), sizes.size() * sizeof(size_t), hipMemcpyHostToDevice));

   chunkOffsets[0] = 0;
   for (auto c = 1; c < sizes.size(); c++) {
      chunkOffsets[c] += sizes[c - 1] + chunkOffsets[c - 1];
   }
   size_t *dOffsets = NULL;
   ERRCHECK(hipMalloc(&dOffsets, chunkOffsets.size() * sizeof(size_t)));
   ERRCHECK(hipMemcpy(dOffsets, chunkOffsets.data(), chunkOffsets.size() * sizeof(size_t), hipMemcpyHostToDevice));

   // run unpack kernels
   Unpack1<float, float>
      <<<ceil(pack.size() / 256.), 256>>>(dOut, dPack, dSizes, sizes.size(), pack.size() * sizeof(float));
   ERRCHECK(hipPeekAtLastError());
   ERRCHECK(hipMemcpy(out.data(), dOut, out.size() * sizeof(float), hipMemcpyDeviceToHost));
   assert(in == out && "Unpack1 failed");

   // ERRCHECK(hipMemset(dOut, 0, out.size() * sizeof(float)));
   // Unpack1_1<float, float><<<ceil(pack.size() / 256.), 256, sizes.size()>>>(dOut, dPack, dSizes, dOffsets,
   // sizes.size(),
   //                                                                        pack.size() * sizeof(float));
   // ERRCHECK(hipPeekAtLastError());
   // ERRCHECK(hipMemcpy(out.data(), dOut, out.size() * sizeof(float), hipMemcpyDeviceToHost));
   // assert(in == out && "Unpack2 failed");

   ERRCHECK(hipMemset(dOut, 0, out.size() * sizeof(float)));
   Unpack2<float, float>
      <<<ceil(pack.size() / 256.), 256>>>(dOut, dPack, dSizes, sizes.size(), pack.size() * sizeof(float));
   ERRCHECK(hipPeekAtLastError());
   ERRCHECK(hipMemcpy(out.data(), dOut, out.size() * sizeof(float), hipMemcpyDeviceToHost));
   assert(in == out && "Unpack2 failed");

   ERRCHECK(hipMemset(dOut, 0, out.size() * sizeof(float)));
   Unpack2_1<float, float><<<ceil(pack.size() / 256.), 256, sizes.size()>>>(dOut, dPack, dSizes, dOffsets, sizes.size(),
                                                                            pack.size() * sizeof(float));
   ERRCHECK(hipPeekAtLastError());
   ERRCHECK(hipMemcpy(out.data(), dOut, out.size() * sizeof(float), hipMemcpyDeviceToHost));
   assert(in == out && "Unpack2.1 failed");

   ERRCHECK(hipMemset(dOut, 0, out.size() * sizeof(float)));
   Unpack3<float, float>
      <<<ceil(pack.size() / 256.), 256>>>(dOut, dPack, dSizes, sizes.size(), pack.size() * sizeof(float));
   ERRCHECK(hipPeekAtLastError());
   ERRCHECK(hipMemcpy(out.data(), dOut, out.size() * sizeof(float), hipMemcpyDeviceToHost));
   assert(in == out && "Unpack3 failed");

   return 0;
}
